// matrix_multiplication.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512  // Matrix size N x N

__global__ void matMulKernel(float* A, float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < n && col < n) {
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // Initialize input matrices
    for (int i = 0; i < N * N; i++) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy input data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    // Launch kernel
    matMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print a sample result
    printf("C[0][0] = %f\n", C[0]);

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(A); free(B); free(C);

    return 0;
}
